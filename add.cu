
#include <hip/hip_runtime.h>
#include <stdio.h>
// Kernel definition
//adds two vectors A and B of size N and stores the result into vector C: 

__global__ void VecAdd(float* A, float* B, float *C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i] ;
}

int main()
{
    // here only 1024 is the maximum number i can use for carrying out the 
    // additions that is the max number of threads i can have per thread block

    int N = 1024;

    float *A, *B, *C, *d_A, *d_B, *d_C;
  	A = (float*)malloc(N*sizeof(float));
  	B = (float*)malloc(N*sizeof(float));
    C = (float*)malloc(N*sizeof(float));


  	hipMalloc(&d_A, N*sizeof(float)); 
  	hipMalloc(&d_B, N*sizeof(float));
    hipMalloc(&d_C, N*sizeof(float));

  	for (int i = 0; i < N; i++) {
    A[i] = 1.0f;
    B[i] = 2.0f;
    C[i] = 0.0f;
  	}

  	hipMemcpy(d_A, A, N*sizeof(float), hipMemcpyHostToDevice);
  	hipMemcpy(d_B, B, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, N*sizeof(float), hipMemcpyHostToDevice);

    // Kernel invocation with N threads
    VecAdd<<<1, N>>>(d_A, d_B, d_C);

    
    hipMemcpy(C, d_C, N*sizeof(float), hipMemcpyDeviceToHost);

    float sum = 0.0f;
    for (int i = 0; i < N; i++){
      sum = sum + C[i];
    }
    printf("Sum is: %f\n", sum); 

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(A);
  free(B);
  free(C);
    
}