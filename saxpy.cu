#include "hip/hip_runtime.h"
#include <stdio.h>
// the kernels such as saxpy are defined using __global__ declaration
// specifier. 
// variables defined with in the device code do not need to be specified as 
// device variables as they are presumed to reside on the device
// in this case the variables n a and i are stored by each thread on 
// registers. and the pointers x and y are pointers to the device memory 
// the first two arguments n and a are not explicitly transfered to device in
// host code this is because function arguments are passed by value by default
// in c or c++ and cuda can automatically handle the transfer of the values

// as we know the kernel is excecuted by multiple threads in parallel.
// but if we want each thread to process a element of resultant array then 
// we need a means of distinguishing each and identifying each thread.
// cuda defines blockdim, blockidx and threadidx. these predefined variables // are of the type dim3.

// the predefined variable blockdim contains the dimension of each thread 
// block

// the predefined variables thread idx and the blockidx contain the index of // thread with in thread block and the thread block with in the grid

// so the index i is the global index that is used to access the elements of // the arrays. 

// griddim is another parameter that stores the dimensions of the grid

// the second line performs the saxpy operation element wisse


__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;

  // the pointers x and y point to the host arrays allocated with malloc in // the typical fashion
  // the arrats dx and dy point to device arrays allocated with cuda malloc
  // the host and device in cuda has different memory spaces, both of which // can be managed from host code 
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  // the host code that inits the host arrays 
  // x is array of ones and y is an array of 2s

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // to init the device arrays copy the data from x,y to dx and dy
  // using cudamemcpy. the fourth argument indicates the direction of copy

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  // the info between the triple chevrons dictates the number of device 
  // threads excecute the kernel in parallel. In cuda programming we call the 
  // group of thread blocks launched are called grid of thread blocks

  // the first argument specifies the number of thread blocks in the grid
  // the second argument specifies the number of threads in the block

  // thread blocks and grids can be made one, two and three dimensional by 
  // passing dim3 values for these arguments
  // this example we neeed one dimension so only integers are passed
  // so this case have 256 threads and use the arithmetic division to 
  // determine the number of blocks 

  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  // after running the kernel, to get results back we copy the results from 
  // device array dy to y using the cudamemxcpy
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  // after done processing, we have to free the memory allocated. for device // memory allocated using cudamalloc use cudafree and use free for host memory

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}


// compiling and running the code. we use the cuda c nvcc compiler to compile // the code in a file with .cu extension
nvcc -o saxpy saxpy.cu